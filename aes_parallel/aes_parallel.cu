#include "hip/hip_runtime.h"
#define DEBUG 0

#include "aes.h"

static double encrypt_file(char* outfile, char* infile, uint8_t* key);
static void __host__ phex(uint8_t* str);

uint8_t key[16] = { (uint8_t)0x2b, (uint8_t)0x7e, (uint8_t)0x15, (uint8_t)0x16,
(uint8_t)0x28, (uint8_t)0xae, (uint8_t)0xd2, (uint8_t)0xa6,
(uint8_t)0xab, (uint8_t)0xf7, (uint8_t)0x15, (uint8_t)0x88,
(uint8_t)0x09, (uint8_t)0xcf, (uint8_t)0x4f, (uint8_t)0x3c };

// The array that stores the round keys.
uint8_t h_roundKey[176];

bool silent = 0;

void print_usage() {
	printf("Usage: aes_parallel.exe <input file> <output file> [<number of runs>]\n");
	return;
}

int main(int argc, char *argv[]) {
	if (argc < 3 || argc > 4) {
		print_usage();
		return 1;
	}

	double cpu_time_used;
	
	if (argc == 3) {
		cpu_time_used = encrypt_file(argv[1], argv[2], key);
		printf("Execution time: %f seconds\n", cpu_time_used);
	}
	else if (argc == 4) {
		uint16_t number_of_runs = atoi(argv[3]);
		if (!number_of_runs) {
			print_usage();
			return 1;
		}

		silent = 1;
		double total_cpu_time_used = 0.0;
		uint8_t i;
		for (i = 0; i < number_of_runs; i++) {
			cpu_time_used = encrypt_file(argv[1], argv[2], key);
			printf("[Run %d] Execution time: %f seconds\n", i, cpu_time_used);
			total_cpu_time_used += cpu_time_used;
		}

		double average_cpu_time_used = total_cpu_time_used / number_of_runs;
		printf("Total execution time: %f seconds\n", total_cpu_time_used);
		printf("Average execution time: %f seconds\n", average_cpu_time_used);
	}

	return 0;
}

double encrypt_file(char* infile, char* outfile, uint8_t* key) {
	uintmax_t plaintext_blocks = 0;
	FILE *fp_in;
	FILE *fp_out;
	hipError_t cudaStatus;

#if defined(DEBUG) && DEBUG
	uint8_t i;
#endif

	fp_in = fopen(infile, "rb");
	if (fp_in == NULL && !silent) {
		fprintf(stderr, "Can't open input file %s!\n", infile);
		exit(1);
	}
	fp_out = fopen(outfile, "wb+");
	if (fp_out == NULL && !silent) {
		fprintf(stderr, "Can't open output file %s!\n", outfile);
		exit(1);
	}

	KeyExpansion(key);
	
#if defined(DEBUG) && DEBUG
	printf("Round Keys:\n");
	for (i = 0; i < ROUNDS + 1; i++) {
		phex(h_roundKey + (i * BLOCKSIZE));
	}
#endif

	// determine size of file, read file into h_plaintext and determine number of plaintext blocks
	fseek(fp_in, 0, SEEK_END);
	uintmax_t plaintext_size = ftell(fp_in);
	rewind(fp_in);
	uint8_t* h_plaintext = (uint8_t*)malloc(plaintext_size);
	uintmax_t bytes_read = fread(h_plaintext, sizeof(uint8_t), plaintext_size, fp_in);
	assert(bytes_read == plaintext_size);
	plaintext_blocks = (bytes_read + BLOCKSIZE - 1) / BLOCKSIZE;
	uint8_t* h_ciphertext = (uint8_t*)malloc(plaintext_blocks*BLOCKSIZE);

	if (!silent) {
		printf("Encrypting file \"%s\"\n", infile);
		printf("File size: %llu bytes\n", plaintext_size);
		printf("Number of plaintext blocks: %llu (blocksize: %d bytes)\n", plaintext_blocks, BLOCKSIZE);
	}

#if defined(DEBUG) && DEBUG
	printf("Plaintext:\n");
	for (i = 0; i < plaintext_blocks; i++) {
		phex(h_plaintext + (i * BLOCKSIZE));
	}
#endif

	// copy h_plaintext and h_roundKey into global device memory
	uint8_t* d_plaintext;
	cudaStatus = hipMalloc((void**)&d_plaintext, sizeof(uint8_t) * (plaintext_blocks * BLOCKSIZE)); // TODO if last block is smaller than BLOCKSIZE, the block maybe needs to be initialized with zero bits, test if this has to be done
	if (cudaStatus != hipSuccess && !silent) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// make sure the last block is padded with zero bytes by initializing the full array with zero bytes
	cudaStatus = hipMemset(d_plaintext, 0, sizeof(uint8_t) * (plaintext_blocks * BLOCKSIZE));
	if (cudaStatus != hipSuccess && !silent) {
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_plaintext, h_plaintext, sizeof(uint8_t)*plaintext_size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess && !silent) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	uint8_t* d_roundKey;
	hipMalloc((void**)&d_roundKey, sizeof(uint8_t)*BLOCKSIZE*(ROUNDS+1));
	if (cudaStatus != hipSuccess && !silent) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	hipMemcpy(d_roundKey, h_roundKey, sizeof(uint8_t)*BLOCKSIZE*(ROUNDS + 1), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess && !silent) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// allocate space for the ciphertext on the device
	uint8_t* d_ciphertext;
	cudaStatus = hipMalloc((void**)&d_ciphertext, sizeof(uint8_t) * (plaintext_blocks * BLOCKSIZE));
	if (cudaStatus != hipSuccess && !silent) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// initialize the ciphertext with all zero // this is not necessary, it seems
	//cudaStatus = hipMemset(d_ciphertext, 0, sizeof(uint8_t) * (plaintext_blocks * BLOCKSIZE));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemset failed!");
	//	goto Error;
	//}

	uintmax_t threads_per_block = THREADS_PER_BLOCK;
	uintmax_t number_of_blocks = (plaintext_blocks + threads_per_block - 1) / threads_per_block;
	uintmax_t shared_memory_size = BLOCKSIZE * THREADS_PER_BLOCK + BLOCKSIZE * (ROUNDS + 1);

	if (!silent) {
		printf("Launching kernel with configuration:\n");
		printf("Threads per block: %lld\n", threads_per_block);
		printf("Number of blocks: %lld\n", number_of_blocks);
		printf("Shared memory size: %lld\n", shared_memory_size);
	}

	// reset last error
	hipGetLastError();

	// measure time
	clock_t start, end;
	double cpu_time_used;

	start = clock();

	cuda_encrypt_block<<<number_of_blocks, threads_per_block/*,shared_memory_size*/>>>(d_ciphertext, d_plaintext, d_roundKey, plaintext_blocks);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess && !silent) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess && !silent) {
		fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	end = clock();
	cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;

	// Copy ciphertext array from device memory to host memory.
	cudaStatus = hipMemcpy(h_ciphertext, d_ciphertext, sizeof(uint8_t) * (plaintext_blocks * BLOCKSIZE), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess && !silent) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	
#if defined(DEBUG) && DEBUG
	printf("Ciphertext after kernel returned:\n");
	for (i = 0; i < plaintext_blocks; i++) {
		phex(h_ciphertext + (i * BLOCKSIZE));
	}
#endif

	// write ciphertext to output file
	fwrite(h_ciphertext, sizeof(uint8_t), BLOCKSIZE * plaintext_blocks, fp_out);

	if (!silent)
		printf("\nEncryption of %llu plaintext blocks successful!\n", plaintext_blocks);
	
	return cpu_time_used;

Error:
	free(h_plaintext);
	free(h_ciphertext);
	free(h_roundKey);

	hipFree(d_plaintext);
	hipFree(d_ciphertext);
	hipFree(d_roundKey);
	
	fclose(fp_in);
	fclose(fp_out);
	exit(1);	
}


// This function produces (ROUNDS+1) round keys. The round keys are used in each round to decrypt the states. 
void KeyExpansion(uint8_t* key) {
	uint32_t i, j, k;
	uint8_t tempa[4]; // Used for the column/row operations

	// The first round key is the key
	for (i = 0; i < KEYWORDS; ++i)
	{
		h_roundKey[(i * 4) + 0] = key[(i * 4) + 0];
		h_roundKey[(i * 4) + 1] = key[(i * 4) + 1];
		h_roundKey[(i * 4) + 2] = key[(i * 4) + 2];
		h_roundKey[(i * 4) + 3] = key[(i * 4) + 3];
	}

	// All other round keys are found from the previous round keys.
	for (; (i < (LANESIZE * (ROUNDS + 1))); ++i)
	{
		for (j = 0; j < 4; ++j)
		{
			tempa[j] = h_roundKey[(i - 1) * 4 + j];
		}
		if (i % KEYWORDS == 0)
		{
			// This function rotates the 4 bytes in a word to the left once.
			// [a0,a1,a2,a3] becomes [a1,a2,a3,a0]

			// Function RotWord()
			{
				k = tempa[0];
				tempa[0] = tempa[1];
				tempa[1] = tempa[2];
				tempa[2] = tempa[3];
				tempa[3] = k;
			}

			// SubWord() is a function that takes a four-byte input word and 
			// applies the S-box to each of the four bytes to produce an output word.

			// Function Subword()
	  {
		  tempa[0] = sbox[tempa[0]];
		  tempa[1] = sbox[tempa[1]];
		  tempa[2] = sbox[tempa[2]];
		  tempa[3] = sbox[tempa[3]];
	  }

			tempa[0] = tempa[0] ^ Rcon[i / KEYWORDS];
		}
		else if (KEYWORDS > 6 && i % KEYWORDS == 4)
		{
			// Function Subword()
			{
				tempa[0] = sbox[tempa[0]];
				tempa[1] = sbox[tempa[1]];
				tempa[2] = sbox[tempa[2]];
				tempa[3] = sbox[tempa[3]];
			}
		}
		h_roundKey[i * 4 + 0] = h_roundKey[(i - KEYWORDS) * 4 + 0] ^ tempa[0];
		h_roundKey[i * 4 + 1] = h_roundKey[(i - KEYWORDS) * 4 + 1] ^ tempa[1];
		h_roundKey[i * 4 + 2] = h_roundKey[(i - KEYWORDS) * 4 + 2] ^ tempa[2];
		h_roundKey[i * 4 + 3] = h_roundKey[(i - KEYWORDS) * 4 + 3] ^ tempa[3];
	}
}


// prints string as hex
static void phex(uint8_t* str) {
	unsigned char i;
	for (i = 0; i < 16; ++i)
		printf("%.2x", str[i]);
	printf("\n");
}